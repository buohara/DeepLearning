#include "hip/hip_runtime.h"
#include "nnkernels.cuh"

__global__ 
void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__
void hadamard(double *a, double *b)
{
    int i = threadIdx.x;
    a[i] = a[i] * b[i];
}

__global__
void sigmoid(double *zVec, double *activations, double *sps)
{
    int i = threadIdx.x;

    double sig = 1.0 / (1.0 + exp(-zVec[i]));
    activations[i] = sig;
    sps[i] = sig * (1.0 - sig);
}

void had(double *a, double *b, int numElements)
{
    hadamard <<< 1, numElements >>> (a, b);
    return;
}

void sigmoids(double *zVec, double *activations, double *sps, int numElements)
{
    sigmoid <<< 1, numElements >>> (zVec, activations, sps);
    return;
}

void addCUDA(
    vector<int> &c,
    vector<int> &a,
    vector<int> &b
)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    
    size_t size = c.size();
    
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);

    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));

    cudaStatus = hipMemcpy(dev_a, a.data(), size * sizeof(int), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_b, b.data(), size * sizeof(int), hipMemcpyHostToDevice);

    addKernel <<< 1, size >>>(dev_c, dev_a, dev_b);
    cudaStatus = hipDeviceSynchronize();

    cudaStatus = hipMemcpy(c.data(), dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}